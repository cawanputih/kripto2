#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <math.h>

typedef unsigned long ulint;
typedef unsigned long long ulint64;

int banyakdata = 1024;
int dimensigrid = 8;
int dimensiblok = 128;

__device__ void modexp(ulint a, ulint b, ulint c, ulint* res) {
	ulint64 s = a;
	ulint64 ans = 1;
	while (b != 0) {
		if (b % 2 == 1) {
			ans = ans * s % c;
			b--;
		}
		b /= 2;
		if (b != 0) {
			s = s * s %c;
		}
	}
	*res = ans;
}

__global__ void kernelenk(ulint *m, ulint e, ulint n, ulint *res) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	modexp(m[i], e, n, res + i);
}

__global__ void kerneldek(ulint *c, ulint d, ulint n, ulint *res) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	modexp(c[i], d, n, res + i);
}

void enkripsiCUDA(ulint *m, ulint e, ulint n, ulint *res) {
	hipSetDevice(0);

	ulint *devm, *devres;

	hipMalloc((void**)&devm, banyakdata * sizeof(ulint));
	hipMalloc((void**)&devres, banyakdata * sizeof(ulint));
	
	hipMemcpy((devm), m, (sizeof(ulint) * banyakdata), hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

	kernelenk << <dimensigrid, dimensiblok>> >(devm,e,n,devres);

	hipEventRecord(stop);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("\nDurasi enkripsi= %f ms\n", milliseconds);


	hipDeviceSynchronize();

	//	COPY FROM DEVICE TO HOST HERE 
	hipMemcpy(res, devres, (sizeof(ulint) * banyakdata), hipMemcpyDeviceToHost);
	
	hipFree(devm);
	hipFree(devres);
}

void dekripsiCUDA(ulint *c, ulint d, ulint n, ulint *res2) {
	hipSetDevice(0);

	//=====================BAGIAN M[] K[] DAN RES[] ====================================//
	ulint *devc, *devres2;
	

	hipMalloc((void**)&devc, banyakdata * sizeof(ulint));
	hipMalloc((void**)&devres2, banyakdata * sizeof(ulint));
	
	hipMemcpy((devc), c, (sizeof(ulint) * banyakdata), hipMemcpyHostToDevice);

	// printf("<<<<<<<<<<<<<<<<<<KERNEL>>>>>>>>>>>>>>>>>\n");

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

	kerneldek << <dimensigrid, dimensiblok>> >(devc,d,n,devres2);

	hipEventRecord(stop);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("\nDurasi enkripsi= %f ms\n", milliseconds);

	hipDeviceSynchronize();

	//	COPY FROM DEVICE TO HOST HERE 
	hipMemcpy(res2, devres2, (sizeof(ulint) * banyakdata), hipMemcpyDeviceToHost);
	
	hipFree(devc);
	hipFree(devres2);

}

void initenkripsi(ulint *m){
	srand(2018);
	for (int i = 0; i < banyakdata; i++) {
		m[i] = rand() % 256;
	}	
}

int main(){
	ulint *m, e, d, n, *res, *res2;

	m = (ulint*)malloc(banyakdata * sizeof(ulint));
	res = (ulint*)malloc(banyakdata * sizeof(ulint));
	res2 = (ulint*)malloc(banyakdata * sizeof(ulint));

	e = 211;
	d = 259;
	n = 299;

	initenkripsi(m);

	printf("<<<<<<<<<<<<<<Pesan Asli>>>>>>>>>>>>>>>\n");
	for (int i = 0; i < 4; i++) {
		printf("m[%d] = %lu\n", i, m[i]);
	}

	printf("m[...]\n");
	printf("m[%d] = %lu\n", banyakdata-1, m[banyakdata-1]);

	enkripsiCUDA(m,e,n,res);

	printf("<<<<<<<<<<<<<<Hasil Enkripsi>>>>>>>>>>>>>>>\n");
	for (int i = 0; i < 4; i++) {
		printf("c[%d] = %lu 	c[%d] = %lu\n", 2*i, res[2*i], 2*i+1, res[2*i+1]);
	}

	printf("c ...\n");
	printf("c[%d] = %lu 	c[%d] = %lu\n", banyakdata * 2-2, res[banyakdata * 2-2], banyakdata *2-1,res[banyakdata*2-1]);

	dekripsiCUDA(res,d,n,res2);

	printf("<<<<<<<<<<<<<<Hasil Dekripsi>>>>>>>>>>>>>>>\n");
	for (int i = 0; i < 4; i++) {
		printf("m[%d] = %lu\n", i, res2[i]);
	}

	printf("m[...]\n");
	printf("m[%d] = %lu\n", banyakdata-1, res2[banyakdata-1]);

	free(m);
	free(res);
	free(res2);

	return 0;
}